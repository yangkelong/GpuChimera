#include "hip/hip_runtime.h"
// ======================================================================== //
// Copyright 2018-2023 Ingo Wald                                            //
//                                                                          //
// Licensed under the Apache License, Version 2.0 (the "License");          //
// you may not use this file except in compliance with the License.         //
// You may obtain a copy of the License at                                  //
//                                                                          //
//     http://www.apache.org/licenses/LICENSE-2.0                           //
//                                                                          //
// Unless required by applicable law or agreed to in writing, software      //
// distributed under the License is distributed on an "AS IS" BASIS,        //
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. //
// See the License for the specific language governing permissions and      //
// limitations under the License.                                           //
// ======================================================================== //
#pragma once
#include "cukd/builder.h"
#include "cukd/fcp.h"  // fcp = "find closest point" query
#include <queue>
#include <iomanip>
#include "ToWallDistance.h"
#include <numeric>

using mydata3 = float3;
//using mydata3 = double3;
using mydata = typename cukd::scalar_type_of<mydata3>::type;

struct MyPoint { 
  mydata3  position;
  // 1 byte for split dimension
  uint8_t split_dim; 
};

struct MyPoint_traits : public cukd::default_data_traits<mydata3>{
  using point_t = mydata3;

  static inline __device__ __host__
  mydata3 get_point(const MyPoint &data)
  { return data.position; }

  static inline __device__ __host__
  mydata  get_coord(const MyPoint &data, int dim)
  { return cukd::get_coord(get_point(data),dim); }

  enum { has_explicit_dim = true };

  static inline __device__ void set_dim(MyPoint &p, int dim){p.split_dim = dim; }

  static inline __device__ int  get_dim(const MyPoint &p) { return p.split_dim; }
};

__global__ void d_fcp_mypoint(mydata *d_results, mydata3 *d_queries, int numQueries,
                      const cukd::box_t<mydata3> *d_bounds, MyPoint *d_nodes,
                      int numNodes, mydata cutOffRadius, int *d_records){
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid >= numQueries)
    return;
  mydata3 queryPos = d_queries[tid];  // 查询点 坐标
  cukd::FcpSearchParams params;
  params.cutOffRadius = cutOffRadius;
  int traverse_node_num = 0;
  int closestID = cukd::cct::fcp<MyPoint, MyPoint_traits>(queryPos, *d_bounds, d_nodes, numNodes, params, d_records[tid]);
  // d_records[tid] = traverse_node_num;
  d_results[tid] = (closestID < 0)
                       ? INFINITY
                       : cukd::distance(queryPos, d_nodes[closestID].position);
}


MyPoint* uploadMyPoints(Point *coords, int N) {
    MyPoint* d_points = 0;
    hipMallocManaged((char**)&d_points, N * sizeof(*d_points));
    if (!d_points)
        throw std::runtime_error("could not allocate points mem...");
    for (int i = 0; i < N; i++) {
        d_points[i].position.x = (mydata) coords[i].x;
        d_points[i].position.y = (mydata) coords[i].y;
        d_points[i].position.z = (mydata) coords[i].z;
    }
    return d_points;
}


template<typename T3>
T3* uploadPoints(Point *coords, int N) {
    T3* d_points = 0;
    hipMallocManaged((char**)&d_points, N * sizeof(*d_points));
    if (!d_points)
        throw std::runtime_error("could not allocate points mem...");
    for (int i = 0; i < N; i++) {
        d_points[i].x = (mydata) coords[i].x;
        d_points[i].y = (mydata) coords[i].y;
        d_points[i].z = (mydata) coords[i].z;
    }
    return d_points;
}

// 查询host函数, cct
__global__ void d_fcp(mydata *d_results, mydata3 *d_queries, int numQueries,
                      /*! the world bounding box computed by the builder */
                      const cukd::box_t<mydata3> *d_bounds, mydata3 *d_nodes,
                      int numNodes, mydata cutOffRadius, int *d_records){
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid >= numQueries)
    return;
  mydata3 queryPos = d_queries[tid];  // 查询点 坐标
  cukd::FcpSearchParams params;
  params.cutOffRadius = cutOffRadius;
  int traverse_node_num = 0;
  int closestID = cukd::cct::fcp(queryPos, *d_bounds, d_nodes, numNodes, params, d_records[tid]);
  // d_records[tid] = traverse_node_num;
  d_results[tid] = (closestID < 0)
                       ? INFINITY
                       : cukd::distance(queryPos, d_nodes[closestID]);
}

__global__ void d_fcp_stackBased(mydata *d_results, mydata3 *d_queries, int numQueries,
                      /*! the world bounding box computed by the builder */
                      const cukd::box_t<mydata3> *d_bounds, mydata3 *d_nodes,
                      int numNodes, mydata cutOffRadius, int *d_records){
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid >= numQueries)
    return;
  mydata3 queryPos = d_queries[tid];  // 查询点 坐标
  cukd::FcpSearchParams params;
  params.cutOffRadius = cutOffRadius;
  int traverse_node_num = 0;
  int closestID = cukd::stackBased::fcp(queryPos, d_nodes, numNodes, params, d_records[tid]);
  // d_records[tid] = traverse_node_num;
  d_results[tid] = (closestID < 0)
                       ? INFINITY
                       : cukd::distance(queryPos, d_nodes[closestID]);
}


// 后续考虑将构建 查询 分开？
void calWalDist();
extern "C"  void toWallDistance(Point* coords, unsigned int n, Point* query_coords, unsigned int numQueries,
    double* result_coords) {
    std::cout << "toWallDistance ..." <<"k-d tree node_num: "<< n<<", query points: "<< numQueries << std::endl;
    using namespace cukd::common;
    cukd::box_t<mydata3>* d_bounds;
    mydata3* d_points = uploadPoints<mydata3>(coords, n);
    hipMallocManaged((void**)&d_bounds, sizeof(cukd::box_t<mydata3>));
    std::cout << "allocated memory for the world space bounding box ..." << std::endl;
    // ==================================================================
    // build the tree.
    // ==================================================================
    std::cout << "calling builder..." << std::endl;
    double t0 = cukd::common::getCurrentTime();
    cukd::buildTree(d_points, n, d_bounds);
    CUKD_CUDA_SYNC_CHECK();
    double t1 = cukd::common::getCurrentTime();
    std::cout << "done building tree, took "
        << cukd::common::prettyDouble(t1 - t0) << "s" << std::endl;
    // 搜索时的最大半径
    mydata cutOffRadius = std::numeric_limits<mydata>::infinity();
    mydata3* d_queries = uploadPoints<mydata3>(query_coords, numQueries);
    // allocate memory for the results
    mydata* d_results;
    CUKD_CUDA_CALL(MallocManaged((void**)&d_results, numQueries * sizeof(*d_results)));
    // ==================================================================
    // do queryies 
    // ==================================================================
    // 记录每个查询点 遍历 tree 时 访问的节点数目
    int *d_records;
    hipMallocManaged((char **)&d_records, numQueries * sizeof(int));
    {
        for (int i = 0; i < numQueries; i++){
            d_records[i] = 0;
        }
        t0 = cukd::common::getCurrentTime();
        int bs = 128;
        int nb = cukd::divRoundUp((int)numQueries, bs);
        d_fcp << <nb, bs >> > (d_results, d_queries, numQueries, d_bounds, d_points, n, cutOffRadius, d_records);
        hipDeviceSynchronize();
        CUKD_CUDA_SYNC_CHECK();
        t1 = cukd::common::getCurrentTime();
        std::cout << "done "
            << " iterations of " << numQueries
            << " fcp queries, took " << cukd::common::prettyDouble(t1 - t0)
            << "s" << std::endl;
        std::cout << "that is " << cukd::common::prettyDouble(numQueries / (t1 - t0))
            << " queries/s" << std::endl;
        double avg_per_query = std::accumulate(d_records, d_records+numQueries, 0.)/numQueries;
        std::cout << "average traverse_node_num per query: " << avg_per_query << std::endl; 
        for (int i = 0; i < numQueries; i++) {
            result_coords[i] = (double) d_results[i];
        }
    }
    {
        for (int i = 0; i < numQueries; i++){
            d_records[i] = 0;
        }
        t0 = cukd::common::getCurrentTime();
        int bs = 128;
        int nb = cukd::divRoundUp((int)numQueries, bs);
        d_fcp_stackBased << <nb, bs >> > (d_results, d_queries, numQueries, d_bounds, d_points, n, cutOffRadius, d_records);
        hipDeviceSynchronize();
        CUKD_CUDA_SYNC_CHECK();
        t1 = cukd::common::getCurrentTime();
        std::cout << "done "
            << " iterations of " << numQueries
            << " fcp queries, took " << cukd::common::prettyDouble(t1 - t0)
            << "s" << std::endl;
        std::cout << "that is " << cukd::common::prettyDouble(numQueries / (t1 - t0))
            << " queries/s" << std::endl;
        double avg_per_query = std::accumulate(d_records, d_records+numQueries, 0.)/numQueries;
        std::cout << "average traverse_node_num per query: " << avg_per_query << std::endl; 
        for (int i = 0; i < numQueries; i++) {
            result_coords[i] = (double) d_results[i];
        }
    }

  { 
    //
    cukd::box_t<mydata3> *d_bounds;
    hipMallocManaged((void **)&d_bounds, sizeof(cukd::box_t<mydata3>));
    MyPoint *d_points = uploadMyPoints(coords, n);
    cukd::buildTree<MyPoint, MyPoint_traits>(d_points, n, d_bounds);
    for (int i = 0; i < numQueries; i++){
      d_records[i] = 0;
    }
    t0 = cukd::common::getCurrentTime();
    int bs = 128;
    int nb = cukd::divRoundUp((int)numQueries, bs);
    d_fcp_mypoint << <nb, bs >> > (d_results, d_queries, numQueries, d_bounds, d_points, n, cutOffRadius, d_records);
    hipDeviceSynchronize();
    CUKD_CUDA_SYNC_CHECK();
    t1 = cukd::common::getCurrentTime();
    std::cout << "done "
        << " iterations of " << numQueries
        << " fcp queries, took " << cukd::common::prettyDouble(t1 - t0)
        << "s" << std::endl;
    std::cout << "that is " << cukd::common::prettyDouble(numQueries / (t1 - t0))
        << " queries/s" << std::endl;
    double avg_per_query = std::accumulate(d_records, d_records+numQueries, 0.)/numQueries;
    std::cout << "average traverse_node_num per query: " << avg_per_query << std::endl; 
    for (int i = 0; i < numQueries; i++) {
        result_coords[i] = (double) d_results[i];
    }
  }

    hipFree(d_points);
    hipFree(d_bounds);
    hipFree(d_queries);
    hipFree(d_results);
}

