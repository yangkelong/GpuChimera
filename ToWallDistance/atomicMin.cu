#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdio> // 用于 printf

// 定义 atomicMin 函数
inline __device__
float atomicMin(float *addr, float value) {
    float old = *addr, assumed;
    if (old <= value) return old;
    do {
        assumed = old;
        old = __int_as_float(atomicCAS((unsigned int*)addr, __float_as_int(assumed), __float_as_int(value)));
        value = min(value, old);
    } while (old != assumed);
    return old;
}

// CUDA 内核函数
__global__ void updateMinKernel(float *globalMin, float *localMins, int size) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < size) {
        // 使用 atomicMin 函数更新全局最小值
        atomicMin(globalMin, localMins[idx]);
    }
}

int main() {
    // 分配设备内存
    float *d_globalMin;
    float *d_localMins;
    constexpr size_t size = 1024; // 假设有1024个局部最小值
    hipMalloc(&d_globalMin, sizeof(float));
    hipMalloc(&d_localMins, size * sizeof(float));

    // 初始化全局最小值
    float globalMin = FLT_MAX;
    hipMemcpy(d_globalMin, &globalMin, sizeof(float), hipMemcpyHostToDevice);

    // 初始化局部最小值数组（例如，使用随机值）
    float localMins[size];
    for (int i = 0; i < size; ++i) {
        // localMins[i] = static_cast<float>(i) / size;
        localMins[i] = static_cast<float>(i) + 0.1;
    }
    hipMemcpy(d_localMins, localMins, size * sizeof(float), hipMemcpyHostToDevice);

    // 启动内核
    int threadsPerBlock = 256;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
    updateMinKernel<<<blocksPerGrid, threadsPerBlock>>>(d_globalMin, d_localMins, size);

    // 复制结果回主机
    float h_globalMin;
    hipMemcpy(&h_globalMin, d_globalMin, sizeof(float), hipMemcpyDeviceToHost);

    // 打印结果
    printf("Global minimum: %f\n", h_globalMin);

    // 释放设备内存
    hipFree(d_globalMin);
    hipFree(d_localMins);

    return 0;
}